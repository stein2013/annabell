#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stdlib.h>
#include "sizes.h"
#include "cuPrintf.cu"
#include "gettime.h"

using namespace std;
using namespace sizes;

double cuda_time;
//#define NCols 512

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code),
      file, line);
    if (abort) exit(code);
  }
}

//The macro CUPRINTF is defined for architectures
//with different compute capabilities.
#if __CUDA_ARCH__ < 200     //Compute capability 1.x architectures
#define CUPRINTF cuPrintf
#else                       //Compute capability 2.x architectures
#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
        blockIdx.y*gridDim.x+blockIdx.x,\
        threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
        __VA_ARGS__)
#endif


__global__ void kernel_SparseActiv_blk1(int NRows, int **LkPt,
    int *Nlk, float *in_sign_arr, float *activ_arr)
{
  //float DefaultMinWg = -1;

  if (blockIdx.x<NRows) {
    int nlk = Nlk[blockIdx.x];
    float in_sign = in_sign_arr[blockIdx.x];
    float sum = in_sign;
    int *lk_pt = LkPt[blockIdx.x];
    if (threadIdx.x<nlk) {
      int inr1 = (*(lk_pt+threadIdx.x));
      atomicAdd(&activ_arr[inr1], sum);
    }
  }
}

__global__ void kernel_SparseActiv_blk2(int NRows, int **LkPt,
    int *Nlk, float *in_sign_arr, float *activ_arr)
{
  //float DefaultMinWg = -1;
  int irow = 65536 + blockIdx.x;

  if (irow<NRows) {
    int nlk = Nlk[irow];
    float in_sign = in_sign_arr[irow];
    float sum = in_sign;
    int *lk_pt = LkPt[irow];
    if (threadIdx.x<nlk) {
      int inr1 = (*(lk_pt+threadIdx.x));
      atomicAdd(&activ_arr[inr1], sum);
    }
  }
}

__global__ void kernel_PrintActiv(int arr_size, int NN, float *activ_arr)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  if (index<10) {
    CUPRINTF("cuda: %d\t%f\n", index, activ_arr[index]);
  }
}

int cuda_CopyInputLinks_fn(int Nssm, int *Nnr, int **Nlk, int ***lk_nr,
    int ***lk_nr2, int*** &h_h_lk_nr, int* &dev_Nnr, int** &dev_Nlk,
    int*** &dev_lk_nr, int NN, float* &dev_activ_arr)
{
  int **h_Nlk;
  //int ***h_h_lk_nr;
  int ***h_d_lk_nr;

  gpuErrchk(hipMalloc((void**)&dev_activ_arr, NN*sizeof(float)));
 
  h_Nlk = (int**)malloc(Nssm*sizeof(int*));
  h_h_lk_nr = (int***)malloc(Nssm*sizeof(int**));
  h_d_lk_nr = (int***)malloc(Nssm*sizeof(int**));

  gpuErrchk(hipMalloc( (void**)&dev_Nnr, Nssm*sizeof(int)));
  gpuErrchk(hipMemcpy(dev_Nnr, Nnr, Nssm*sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMalloc( (void**)&dev_Nlk, Nssm*sizeof(int*)));
  gpuErrchk(hipMalloc( (void**)&dev_lk_nr, Nssm*sizeof(int**)));

  for (int issm=0; issm<Nssm; issm++) {
    gpuErrchk(hipMalloc( (void**)&h_Nlk[issm], Nnr[issm]*sizeof(int)));
    gpuErrchk(hipMemcpy(h_Nlk[issm], Nlk[issm], Nnr[issm]*sizeof(int),
      hipMemcpyHostToDevice));
    h_h_lk_nr[issm] = (int**)malloc(Nnr[issm]*sizeof(int*));
    for (int inr=0; inr<Nnr[issm]; inr++) {
      gpuErrchk(hipMalloc( (void**)&h_h_lk_nr[issm][inr],
                       Nlk[issm][inr]*sizeof(int)));
      gpuErrchk(hipMemcpy(h_h_lk_nr[issm][inr], lk_nr[issm][inr],
        Nlk[issm][inr]*sizeof(int), hipMemcpyHostToDevice));
    }
    gpuErrchk(hipMalloc((void**)&h_d_lk_nr[issm], Nnr[issm]*sizeof(int*)));
    gpuErrchk(hipMemcpy(h_d_lk_nr[issm], h_h_lk_nr[issm],
      Nnr[issm]*sizeof(int*), hipMemcpyHostToDevice));
  } 
  gpuErrchk(hipMemcpy(dev_Nlk, h_Nlk, Nssm*sizeof(int*),
    hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dev_lk_nr, h_d_lk_nr, Nssm*sizeof(int**),
    hipMemcpyHostToDevice));

  //for (int issm=0; issm<Nssm; issm++) {
  //  free(h_h_lk_nr[issm]);
  //}
  free(h_Nlk);
  //free(h_h_lk_nr);
  free(h_d_lk_nr);

  cuda_time = 0;
  return 0;
}

int cuda_SparseActiv_fn(int NRows, int **LkPt, int *Nlk,
    float *in_sign_arr, int NN, float *activ_arr, int *dev_Nnr,
    int ***dev_lk_nr, float *dev_activ_arr)
{
  int **dev_LkPt;
  int *dev_Nlk;
  float *dev_in_sign;
  struct timespec clk0, clk1;

  //gpuErrchk( hipDeviceSynchronize() ); GetMonotonicTime(&clk0);
  gpuErrchk(hipMalloc((void**)&dev_LkPt, NRows*sizeof(int*)));
  gpuErrchk(hipMalloc((void**)&dev_Nlk, NRows*sizeof(int)));
  gpuErrchk(hipMalloc( (void**)&dev_in_sign, NRows*sizeof(float)));

  gpuErrchk(hipMemcpy(dev_LkPt, LkPt, NRows*sizeof(int*),
    hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dev_Nlk, Nlk, NRows*sizeof(int),
    hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dev_in_sign, in_sign_arr, NRows*sizeof(float),
    hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(dev_activ_arr, activ_arr, NN*sizeof(float),
    hipMemcpyHostToDevice));

  gpuErrchk( hipDeviceSynchronize() ); GetMonotonicTime(&clk0);
  kernel_SparseActiv_blk1<<< 65535, 512 >>>(NRows, dev_LkPt, dev_Nlk,
    dev_in_sign, dev_activ_arr);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  kernel_SparseActiv_blk2<<< 65535, 512 >>>(NRows, dev_LkPt, dev_Nlk,
    dev_in_sign, dev_activ_arr);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  gpuErrchk( hipDeviceSynchronize() ); GetMonotonicTime(&clk1);

  gpuErrchk(hipMemcpy(activ_arr, dev_activ_arr, NN*sizeof(float),
             hipMemcpyDeviceToHost));

  gpuErrchk(hipFree(dev_LkPt));
  gpuErrchk(hipFree(dev_Nlk));
  gpuErrchk(hipFree(dev_in_sign));

  //gpuErrchk(hipDeviceSynchronize()); GetMonotonicTime(&clk1);

  cuda_time = cuda_time
    + clk1.tv_sec - clk0.tv_sec + (double)(clk1.tv_nsec - clk0.tv_nsec)*1e-9;

  //cout << "cuda time: " << cuda_time << endl;
  //cout << "NRows: " << NRows << "NCols: " << NCols << endl;
  // NCols = 67158 after train_all_cuda.txt
  //exit(0);

  return 0;
}
